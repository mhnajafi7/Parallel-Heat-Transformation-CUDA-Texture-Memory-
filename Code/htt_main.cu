#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "cstdlib"
#include "ctime"
#include "hip/hip_runtime.h"
#include ""
#include "iostream"
#include "gputimer.h"
#include "gpuerrors.h"
#include "htt.h"

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
double calc_mse (float* data1, float* data2, int size);
void cpuKernel(const float* const a,float* c, const int m, const int n);
void gpuKernels(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time);
// =================================================================================

int main(int argc, char** argv) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);
	
	// get parameter from command line to build Matrix dimension
	// check for 10<=m<=13, because m>=14 do not fit in the memory of our GPU, i.e., 1GB.
	int m = atoi(argv[1]);
    int n = (1 << m);
	
	// allocate memory in CPU for calculation
	float* a;
	float* c_serial;
	float* c;
	a        = (float*)malloc(n * sizeof(float));
	c_serial = (float*)malloc(n * sizeof(float));
	c        = (float*)malloc(n * sizeof(float));
	
	// fill a, b matrices with random values between -16.0f and 16.0f
	srand(static_cast<unsigned int>(time(0)));
	fill(a, n);

	// CPU calculations
	clock_t cput1 = clock();
	cpuKernel (a,c_serial, m, n);
	clock_t cput2 = clock();
		
	// GPU calculations
	double gpu_kernel_time = 0.0;
	clock_t t1 = clock(); 
	gpuKernels (a,c, m, n, &gpu_kernel_time);
    clock_t t2 = clock(); 
		
	// check correctness of GPU calculations against CPU
	double mse = 0.0;
	mse += calc_mse( c_serial, c, n );


	printf("m=%d n=%d CPU=%g ms GPU=%g ms GPU-Kernel=%g ms mse=%g\n",
	m, n, (cput2-cput1)/1000.0, (t2-t1)/1000.0, gpu_kernel_time, mse);

	/*for (int i=0; i<n; ++i)
        printf("a=%f c_parallel=%f c_serial=%f\n",a[i],c[i],c_serial[i]);	*/
	// free allocated memory for later use
	free(a);
	free(c_serial);
	free(c);
   
	return 0;
}

void fill(float* data, int size) {

    for (int i=0; i<size; ++i){
        int randomInt = rand();
		float randomFloat = 20.0f +(randomInt / (RAND_MAX + 1.0f)) * 10.0f;
		data[i] = randomFloat;
	}
}

double calc_mse (float* data1, float* data2, int size) {
	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		double e = data1[i]-data2[i];
		e = e * e;
		mse += e;
	}
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel(const float* const a,float* c, const int m, const int n) { // entire matrix 
    for(int i = 0; i < n ; i++){
        float newTemp = a[i];
        if(i==0)
            newTemp += k_const * ( a[i+1] - a[i] );
        else if(i==n-1)
            newTemp += k_const * ( a[i-1] - a[i] );
        else
            newTemp += k_const * ( a[i+1] + a[i-1] - 2 * a[i] );
        c[i] = newTemp;
    }
 
}


//-----------------------------------------------------------------------------
void gpuKernels(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time) {

	float* ad;
	float* cd;


    HANDLE_ERROR(hipMalloc((void**)&ad, n * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(cd, c, n * sizeof(float), hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipBindTexture(NULL, texref, ad, n * sizeof(float)));
	//dim3 dimGrid = getDimGrid(m,n); //modify this function in bmm.cu
	//dim3 dimBlock = getDimBlock(m,n); //modify this function in bmm.cu

	GpuTimer timer;
    timer.Start();
	gpuKernel(ad,cd,n,m);
	//kernelFunc<<< (16),(1024) >>>(ad , cd, n, m); //modify this function in bmm.cu
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
    
	HANDLE_ERROR(hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost));
	//hipUnbindTexture(texref);

    HANDLE_ERROR(hipFree(ad));
    HANDLE_ERROR(hipFree(cd));
}
