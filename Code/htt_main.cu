#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "cstdlib"
#include "ctime"
#include "hip/hip_runtime.h"
#include ""
#include "iostream"
#include "gputimer.h"
#include "gpuerrors.h"
#include "htt.h"

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
double calc_mse (float* data1, float* data2, int size);
void cpuKernel(const float* const a,float* c, float* temp, const int m, const int n);
void gpuKernels(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time);
// =================================================================================

int main(int argc, char** argv) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);
	
	// get parameter from command line to build Matrix dimension
	// check for 10<=m<=13, because m>=14 do not fit in the memory of our GPU, i.e., 1GB.
	int m = atoi(argv[1]);
    int n = (1 << m);
	
	// allocate memory in CPU for calculation
	float* a;
	float* c_serial;
	float* c;
	float* temp;

	a        = (float*)malloc(n*n * sizeof(float));
	c_serial = (float*)malloc(n*n * sizeof(float));
	c        = (float*)malloc(n*n * sizeof(float));
	temp     = (float*)malloc(n*n * sizeof(float));
<<<<<<< HEAD
	
=======
>>>>>>> 86e7e9a2f48372d5e66f5a80cb6a97cfb2bc59f9
	// fill a, b matrices with random values between 20.0f and 30.0f
	srand(0); // If you really want ranodm nubmers, change it like: srand(static_cast<unsigned int>(time(0)));
	fill(a, n*n);

	// CPU calculations
	cpuKernel (a,c_serial,temp, m, n);
		
	// GPU calculations
	double gpu_kernel_time = 0.0;
	clock_t t1 = clock(); 
	gpuKernels (a,c, m, n, &gpu_kernel_time);
    clock_t t2 = clock(); 
		
	// check correctness of GPU calculations against CPU
	double mse = 0.0;
	mse += calc_mse( c_serial, c, n*n );


	printf("m=%d n=%d GPU=%g ms GPU-Kernel=%g ms mse=%g\n",
	m, n, (t2-t1)/1000.0, gpu_kernel_time, mse);
	/*
	for(int i = 0; i < n ; i++){
    	for(int j = 0; j < n ; j++){
        printf("%.1f\t",a[i*n+j]);	
		}
		printf("\n");	
	}
	printf("*************************** \n");

	for(int i = 0; i < n ; i++){
    	for(int j = 0; j < n ; j++){
        printf("%.1f\t",c_serial[i*n+j]);	
		}
		printf("\n");	
	}
	printf("*************************** \n");

	for(int i = 0; i < n ; i++){
    	for(int j = 0; j < n ; j++){
        printf("%.1f\t",c[i*n+j]);	
		}
		printf("\n");	
	}*/
	// free allocated memory for later use
	free(a);
	free(c_serial);
	free(c);
	free(temp);

	return 0;
}

void fill(float* data, int size) {

    for (int i=0; i<size; ++i){
        int randomInt = rand();
		float randomFloat = 20.0f +(randomInt / (RAND_MAX + 1.0f)) * 10.0f;
		data[i] = randomFloat;
	}
}

double calc_mse (float* data1, float* data2, int size) {
	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		double e = data1[i]-data2[i];
		e = e * e;
		mse += e;
	}
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel(const float* const a,float* c, float* temp, const int m, const int n) { // entire matrix
	for(int i = 0; i < n ; i++){
		for(int j = 0; j < n ; j++){
<<<<<<< HEAD
			c[i*n+j] = a[i*n+j];
=======

			float newTemp = a[i*n+j];
			int rt,lt,cr,up,dn;
			
			rt = i*n+(j + 1);	//right
			lt = i*n+(j - 1);	//left
			cr = i*n+j;		//center
			up = (i - 1)*n+j;	//up
			dn = (i + 1)*n+j;	//down
				
			

			if(i==0)	up = cr;
			if(i==n-1)	dn = cr;
			if(j==0)	lt = cr;
			if(j==n-1)	rt = cr;

			
			
			newTemp += k_const * ( a[rt] + a[lt] + a[up] + a[dn] - 4 * newTemp );
			
			c[i*n+j] = newTemp;
		
>>>>>>> 86e7e9a2f48372d5e66f5a80cb6a97cfb2bc59f9
		}
	}
	
	
	for(int count = 0; count <5; count++){

		for(int i = 0; i < n ; i++){
		
			for(int j = 0; j < n ; j++){

				float newt = c[i*n+j];
				int rt,lt,cr,up,dn;
				
				rt = i*n+(j + 1);	//right
				lt = i*n+(j - 1);	//left
				cr = i*n+j;		//center
				up = (i - 1)*n+j;	//up
				dn = (i + 1)*n+j;	//down
					
				if(i==0)	up = cr;
				if(i==n-1)	dn = cr;
				if(j==0)	lt = cr;
				if(j==n-1)	rt = cr;

				newt += k_const * ( c[rt] + c[lt] + c[up] + c[dn] - 4 * c[cr] );
				
				temp[i*n+j] = newt;
			
			}
		}

		for(int i = 0; i < n ; i++){
			for(int j = 0; j < n ; j++){
				c[i*n+j] = temp[i*n+j];
			}
		}
	}


}

//-----------------------------------------------------------------------------
void gpuKernels(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time) {

	float* ad;
	float* cd;


    HANDLE_ERROR(hipMalloc((void**)&ad, n * n * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n * n * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(ad, a, n * n * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(cd, a, n * n * sizeof(float), hipMemcpyHostToDevice));
<<<<<<< HEAD
	// HANDLE_ERROR(hipBindTexture(NULL, texref, ad, n * sizeof(float)));
	//dim3 dimGrid = getDimGrid(m,n); //modify this function in bmm.cu
	//dim3 dimBlock = getDimBlock(m,n); //modify this function in bmm.cu

	GpuTimer timer;
    timer.Start();
	for(int count = 0 ; count <5 ; count++){
		gpuKernel(ad,cd,n,m);
		hipMemcpy(c, cd, n * n * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(ad, c, n * n * sizeof(float), hipMemcpyDeviceToHost);
    }
	//kernelFunc<<< (16),(1024) >>>(ad , cd, n, m); //modify this function in bmm.cu
=======

	GpuTimer timer;
    timer.Start();
	gpuKernel(ad,cd,n,m);
>>>>>>> 86e7e9a2f48372d5e66f5a80cb6a97cfb2bc59f9
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
    
	HANDLE_ERROR(hipMemcpy(c, cd, n * n * sizeof(float), hipMemcpyDeviceToHost));
<<<<<<< HEAD
	//hipUnbindTexture(texref);

=======
	
>>>>>>> 86e7e9a2f48372d5e66f5a80cb6a97cfb2bc59f9
    HANDLE_ERROR(hipFree(ad));
    HANDLE_ERROR(hipFree(cd));
}