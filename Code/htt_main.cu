#include "hip/hip_runtime.h"
//Do NOT MODIFY THIS FILE

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#include "gputimer.h"
#include "gpuerrors.h"
#include "htt.h"

// ===========================> Functions Prototype <===============================
void fill(float* data, int size);
double calc_mse (float* data1, float* data2, int size);
void cpuKernel(const float* const a,float* c, const int m, const int n);
void gpuKernel(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time);
// =================================================================================

int main(int argc, char** argv) {

    struct hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device Name: %s\n", p.name);
	
	// get parameter from command line to build Matrix dimension
	// check for 10<=m<=13, because m>=14 do not fit in the memory of our GPU, i.e., 1GB.
	int m = atoi(argv[1]);
    int n = (1 << m);
	
	// allocate memory in CPU for calculation
	float* a;
	float* c_serial;
	float* c;
	a        = (float*)malloc(n * sizeof(float));
	c_serial = (float*)malloc(n * sizeof(float));
	c        = (float*)malloc(n * sizeof(float));
	
	// fill a, b matrices with random values between -16.0f and 16.0f
	srand(0);
	fill(a, n);

	// CPU calculations
	cpuKernel (a,c_serial, m, n);
		
	// GPU calculations
	double gpu_kernel_time = 0.0;
	clock_t t1 = clock(); 
	gpuKernel (a,c, m, n, &gpu_kernel_time);
    clock_t t2 = clock(); 
		
	// check correctness of GPU calculations against CPU
	double mse = 0.0;
	mse += calc_mse( c_serial, c, n );


	printf("m=%d n=%d GPU=%g ms GPU-Kernel=%g ms mse=%g\n",
	m, n, (t2-t1)/1000.0, gpu_kernel_time, mse);
		
	// free allocated memory for later use
	free(a);
	free(c_serial);
	free(c);
   
	return 0;
}

void fill(float* data, int size) {
    for (int i=0; i<size; ++i)
        data[i] = (float) (rand() % 11 + 20);
}

double calc_mse (float* data1, float* data2, int size) {
	double mse = 0.0;
	int i; for (i=0; i<size; i++) {
		double e = data1[i]-data2[i];
		e = e * e;
		mse += e;
	}
	return mse;
}
//-----------------------------------------------------------------------------
void cpuKernel(const float* const a,float* c, const int m, const int n) { // entire matrix
    for(int i = 0; i < n ; i++){
        float newTemp = a[i];
        if(i==0)
            newTemp += k_const * ( a[i+1] - a[i] );
        else if(i==n-1)
            newTemp += k_const * ( a[i-1] - a[i] );
        else
            newTemp += k_const * ( a[i+1] + a[i-1] - 2 * a[i] );
        c[i] = newTemp;
    }
}


//-----------------------------------------------------------------------------
void gpuKernel(const float* const a, float* c, const int m, const int n, double* gpu_kernel_time) {

	float* ad;
	float* cd;

    HANDLE_ERROR(hipMalloc((void**)&ad, n * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&cd, n * sizeof(float)));

    HANDLE_ERROR(hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice));

	//dim3 dimGrid = getDimGrid(m,n); //modify this function in bmm.cu
	//dim3 dimBlock = getDimBlock(m,n); //modify this function in bmm.cu

	GpuTimer timer;
    timer.Start();
	kernelFunc<<< (16),(1024) >>>(ad , cd, n, m); //modify this function in bmm.cu
	timer.Stop();
	*gpu_kernel_time = timer.Elapsed();
    
	HANDLE_ERROR(hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(ad));
    HANDLE_ERROR(hipFree(cd));
}
