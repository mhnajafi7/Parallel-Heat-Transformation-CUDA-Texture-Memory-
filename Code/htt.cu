#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x

#define tilex 1
#define tiley 1


// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

//-----------------------------------------------------------------------------
__global__ void kernelFunc(const float* oldtemperature,float* newtemperature, const unsigned int N, const unsigned int M)
{

	int x = tx + bx * blockDim.x;
	int offset = x;

	int right = offset + 1;
	int left  = offset - 1;
	if(x == 0)	left++;
	if(x == N - 1)	right--;

	
	newtemperature[offset] = oldtemperature[offset] + k_const * (oldtemperature[left] + oldtemperature[right] - 2 * oldtemperature[offset] );
}

void gpuKernel(const float* ad,float* cd, const unsigned int N, const unsigned int M){

	kernelFunc<<< (16),(1024) >>>(ad , cd, N, M);

}
