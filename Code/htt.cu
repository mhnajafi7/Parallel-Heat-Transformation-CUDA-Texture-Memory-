#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "htt.h"

#define tx threadIdx.x


#define bx blockIdx.x

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
/*dim3 getDimGrid(const int m, const int n) {
        dim3 dimGrid(1);
        return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
        dim3 dimBlock(1024);
        return dimBlock;
}*/
//-----------------------------------------------------------------------------
__global__ void kernelFunc(const float* oldtemperature,float* newtemperature, const unsigned int N, const unsigned int M)
{
	int x = tx + bx * blockDim.x;
	int offset = x;

	int right = offset + 1;
	int left  = offset - 1;

	if(x == 0)	left++;
	if(x == N - 1)	right--;

	newtemperature[offset] = oldtemperature[offset] + k_const * (oldtemperature[left] + oldtemperature[right] - 2 * oldtemperature[offset] );
}
