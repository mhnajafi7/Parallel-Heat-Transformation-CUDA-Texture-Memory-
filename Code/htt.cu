#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x




// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
/*dim3 getDimGrid(const int m, const int n) {
        dim3 dimGrid(1);
        return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
        dim3 dimBlock(1024);
        return dimBlock;
}*/
//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* newtemperature,const float* oldtemperature, const unsigned int N)
{

	int x = tx + bx * blockDim.x;
	int offset = x;

	int right = offset + 1;
	int left  = offset - 1;
	if(x == 0)	left++;
	if(x == N - 1)	right--;

	float r = tex1Dfetch(texref,right);
	float l = tex1Dfetch(texref,left);
	float c = tex1Dfetch(texref,offset);

	// using texture memory
	// newtemperature[offset] = c + k_const * (r + l - 2 * c);

	// linear mode 
	newtemperature[offset] = oldtemperature[offset] + k_const * (oldtemperature[left] + oldtemperature[right] - 2 * oldtemperature[offset] );
}

void gpuKernel(const float* ad,float* cd, const unsigned int N, const unsigned int M){
	hipBindTexture(NULL, texref, ad, N * sizeof(float));
	kernelFunc<<< (4096),(1024) >>>(cd,ad, N);	// for m = 22

}
