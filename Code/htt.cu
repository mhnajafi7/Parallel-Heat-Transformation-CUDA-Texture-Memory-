#include "hip/hip_runtime.h"
#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x
#define ty threadIdx.y
#define by blockIdx.y
#define tl N

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* newtemperature, const float* oldtemperature, const unsigned int N)
{
    int col = tx + bx * tl;
    int row = ty + by * tl;
    int index = row * N + col;

    if (row < N && col < N) {
    newtemperature[index] = oldtemperature[index];
    }
}

void gpuKernel(const float* ad, float* cd, const unsigned int N, const unsigned int M)
{
    dim3 blockSize(tl, tl);  // Adjust block size as needed
    dim3 gridSize((N+tl-1)/tl, (N+tl-1)/tl);

    kernelFunc<<<gridSize, blockSize>>>(cd, ad, N);
}
