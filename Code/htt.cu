#include "hip/hip_runtime.h"
#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x
#define ty threadIdx.y
#define by blockIdx.y

#define cst 16      // Define a constant value

// Define a texture memory
texture<float,hipTextureType1D,hipReadModeElementType> texref;

__global__ void kernelFunc(float* newtemperature, const float* oldtemperature, const unsigned int N)
{   
    // Calculate column and row indices based on thread and block indices
    int col = tx + bx * cst;
    int row = ty + by * cst;
    int index = col + row * blockDim.x * gridDim.x;

    // Calculate indices for neighboring elements
	int left = index - 1;
    int right = index + 1;
    if (col == 0) left++;
    if (col == N-1) right--;

    int top = index - N;
    int bottom = index + N;
    if (row == 0) top += N;
    if (row == N-1) bottom -= N;

    // Fetch values from texture memory for neighboring elements
    float r = tex1Dfetch(texref,right);
	float l = tex1Dfetch(texref,left);
	float c = tex1Dfetch(texref,index);
    float t = tex1Dfetch(texref,top);
    float b = tex1Dfetch(texref,bottom);
    
	// Calculate the new temperature using texture memory
	newtemperature[index] = c + k_const * (r + l + t + b - 4 * c);

	/*
    // Calculate the new temperature not texture memory
    newtemperature[index] = oldtemperature[index] + k_const * (oldtemperature[left] + oldtemperature[right] + oldtemperature[top] + oldtemperature[bottom]- 4 * oldtemperature[index] );
    */

}

void gpuKernel(float* ad, float* cd, const unsigned int N, const unsigned int M)
{

    // Define block size and grid size
    dim3 blockSize(cst, cst);  // Adjust block size as needed
    dim3 gridSize(N/cst, N/cst);

    // Bind the texture to the input data on GPU memory
    hipBindTexture(NULL, texref, ad, N * N * sizeof(float));

    // Launch the CUDA kernel function
    kernelFunc<<<gridSize,blockSize>>>(cd, ad, N);

    // Unbind the texture after kernel execution
    hipUnbindTexture(texref);

}
