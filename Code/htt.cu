#include "hip/hip_runtime.h"
#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x
#define ty threadIdx.y
#define by blockIdx.y
#define tl 16

//-----------------------------------------------------------------------------
__global__ void kernelFunc(float* newtemperature, const float* oldtemperature, const unsigned int N)
{
    int col = tx + bx * tl;
    int row = ty + by * tl;
    int index = col + row * blockDim.x * gridDim.x;


	int left = index - 1;
    int right = index + 1;
    if (col == 0) left++;
    if (col == N-1) right--;

    int top = index - N;
    int bottom = index + N;
    if (row == 0) top += N;
    if (row == N-1) bottom -= N;

    float r = tex1Dfetch(texref,right);
	float l = tex1Dfetch(texref,left);
	float c = tex1Dfetch(texref,index);
    float t = tex1Dfetch(texref,top);
    float b = tex1Dfetch(texref,bottom);
	// using texture memory
	 newtemperature[index] = c + k_const * (r + l + t + b - 4 * c);

	// linear mode 
	//newtemperature[index] = oldtemperature[index] + k_const * (oldtemperature[left] + oldtemperature[right] + oldtemperature[top] + oldtemperature[bottom]- 4 * oldtemperature[index] );

}

void gpuKernel(const float* ad, float* cd, const unsigned int N, const unsigned int M)
{
    dim3 blockSize(tl, tl);  // Adjust block size as needed
    dim3 gridSize((N+tl-1)/tl, (N+tl-1)/tl);

    hipBindTexture(NULL, texref, ad, N * N * sizeof(float));

    kernelFunc<<<gridSize, blockSize>>>(cd, ad, N);
}
