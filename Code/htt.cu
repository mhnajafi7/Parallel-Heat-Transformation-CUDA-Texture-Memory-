#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "htt.h"

#define tx threadIdx.x
#define bx blockIdx.x




// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!
/*dim3 getDimGrid(const int m, const int n) {
        dim3 dimGrid(1);
        return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
        dim3 dimBlock(1024);
        return dimBlock;
}*/
//-----------------------------------------------------------------------------
__global__ void kernelFunc(const float* oldtemperature,float* newtemperature, const unsigned int N, const unsigned int M)
{

	int x = tx + bx * blockDim.x;
	int offset = x;

	int right = offset + 1;
	int left  = offset - 1;
	if(x == 0)	left++;
	if(x == N - 1)	right--;

	float le,ri,ce;
	le = tex1Dfetch(texref,left);
	ri = tex1Dfetch(texref,right);
	ce = tex1Dfetch(texref,offset);

	newtemperature[offset] = ce + k_const * (ri + le - 2 * ce);

	//newtemperature[offset] = oldtemperature[offset] + k_const * (oldtemperature[left] + oldtemperature[right] - 2 * oldtemperature[offset] );
}

void gpuKernel(const float* ad,float* cd, const unsigned int N, const unsigned int M){
	/*HANDLE_ERROR(*/hipBindTexture(NULL, texref, ad, N * sizeof(float))/*)*/;
	kernelFunc<<< (16),(1024) >>>(ad , cd, N, M);

}
